#include <stdio.h>
#include <hip/hip_runtime.h>
int main()	{
	int dev_count;
	hipDeviceProp_t dev_prop;
	hipGetDeviceCount(&dev_count);
	printf("the number of cuda device is %d\n",dev_count);
	hipGetDeviceProperties(&dev_prop,0);
	printf("the number of max threads per block is:%d\n",dev_prop.maxThreadsPerBlock);
	printf("the number of streaming multiprocessors(SM) is:%d\n",dev_prop.multiProcessorCount);
	return 0;
}
