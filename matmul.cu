#include <stdio.h>
#include <hip/hip_runtime.h>
#define BLOCK_SIZE 16
__global__ void matmulKernel(float* mat_in1,float* mat_in2, float* mat_out,int mat_dim);

int main()	{
	float *h_M, *h_N, *h_P, *d_M, *d_N, *d_P;
	int i,width=10;
	int size=width*width*sizeof(float);

	dim3 block_dim(BLOCK_SIZE,BLOCK_SIZE,1);
	int grid_size=width/BLOCK_SIZE;
	if(width%BLOCK_SIZE) grid_size++;
	dim3 grid_dim(grid_size,grid_size,1);

	h_M=(float*)malloc(size);
	h_N=(float*)malloc(size);
	h_P=(float*)malloc(size);
	hipMalloc((void**)&d_M,size);
	hipMalloc((void**)&d_N,size);
	hipMalloc((void**)&d_P,size);

	if(h_M==0||h_N==0||h_P==0||d_M==0||d_N==0||d_P==0)	{
		printf("memory locate fail!\n");
	}

	for(i=0;i<width*width;i++)	{
		h_M[i]=1.2*i;
		h_N[i]=1.4*i;
	}

	hipMemcpy(d_M,h_M,size,hipMemcpyHostToDevice);
	hipMemcpy(d_N,h_N,size,hipMemcpyHostToDevice);
	
	matmulKernel<<<grid_dim,block_dim>>>(d_M,d_N,d_P,width);
	
	hipMemcpy(h_P,d_P,size,hipMemcpyDeviceToHost);

	printf("firt row of the results matrix P:\n");
	for(i=0;i<width;i++)	{
		printf("%f,  ",h_P[i]);
	}
	printf("\n");
	printf("the right answer should be:\n");

	for(i=0;i<width;i++)	{
		float sum=0;
		for(int k=0;k<width;k++)	{
			sum+=h_M[k]*h_N[k*width+i];
		}
		printf("%f,  ",sum);
	}
	printf("\n");

	free(h_M);free(h_N);free(h_P);
	hipFree(d_M);hipFree(d_N);hipFree(d_P);
	return 0;
}

__global__ void matmulKernel(float* mat1,float* mat2, float* matP,int dim)	{
	int thread_x,thread_y,i;
	thread_x=blockIdx.x*blockDim.x+threadIdx.x;
	thread_y=blockIdx.y*blockDim.y+threadIdx.y;
	if(thread_x<dim&&thread_y<dim)	{
		float P_value=0.;
		for(i=0;i<dim;i++)	{
			P_value+=mat1[thread_y*dim+i]*mat2[i*dim+thread_x];
		}
		matP[thread_y*dim+thread_x]=P_value;
	}
}
