#include <stdio.h>
#include <hip/hip_runtime.h>
//be careful the block_size*block_size should not exceed 1024
#define BLOCK_SIZE 32 
__global__ void pictureKernel(float* d_pix,int X, int Y);
int main() {
	float *h_pixin, *h_pixout, *d_pix;
	int x=76,y=62,i;	//2D data size
	int grid_x=x/BLOCK_SIZE,grid_y=y/BLOCK_SIZE;
	int size=x*y*sizeof(float);

	dim3 dim_block(BLOCK_SIZE,BLOCK_SIZE,1);
 	if(x%BLOCK_SIZE) grid_x++;
 	if(y%BLOCK_SIZE) grid_y++;
	dim3 dim_grid(grid_x,grid_y,1);
	printf("grid size is:grid_x=%d,grid_y=%d\n",grid_x,grid_y);

	h_pixin=(float*)malloc(size);
	h_pixout=(float*)malloc(size);
	hipMalloc((void**) &d_pix,size);

	if(h_pixin==NULL||h_pixout==NULL||d_pix==NULL)	{
		printf("malloc failed!\n");
	}

	for(i=0;i<x*y;i++)	{
		h_pixin[i]=i*1.5;
	}

	hipMemcpy(d_pix,h_pixin,size,hipMemcpyHostToDevice);
	pictureKernel<<<dim_grid,dim_block>>>(d_pix,x,y); //after this d_pix changed

  hipError_t error = hipGetLastError();
  if(error != hipSuccess)
  {
    printf("CUDA Error: %s\n", hipGetErrorString(error));
    return 1;
	}

	hipMemcpy(h_pixout,d_pix,size,hipMemcpyDeviceToHost);

	for(i=0;i<x*y;i++)	{
		printf("h_pixin[i]=%f,h_pixout=%f\n",h_pixin[i],h_pixout[i]);
	}
	return 0;
}

__global__ void pictureKernel(float* d_pix,int X, int Y) {
	int thread_x=blockDim.x*blockIdx.x+threadIdx.x;
	int thread_y=blockDim.y*blockIdx.y+threadIdx.y;
//	printf("thread_x=%d,blockDim.x=%d,blockIdx.x=%d,threadIdx=%d\n",thread_x,blockDim.x,blockIdx.x,threadIdx.x);
//	printf("thread_y=%d,blockDim.y=%d,blockIdx.y=%d,threadIdy=%d\n",thread_y,blockDim.y,blockIdx.y,threadIdx.y);
//	use this printf nvcc -arch compute_20 pixel.cu
	if(thread_x<X&&thread_y<Y)	{
		d_pix[thread_y*X+thread_x]*=2;
	}
}
