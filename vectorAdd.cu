#include <stdio.h>
#include <hip/hip_runtime.h>
void vectorAdd(double* A, double* B,double* C,int n);
__global__ void vecAddKernel(double* A, double* B, double* C, int n);
int main()	{
 double *h_A, *h_B, *h_C;
 int i;
 long N=10000;
 int size=N*sizeof(double);
 h_A=(double*)malloc(size);
 h_B=(double*)malloc(size);
 h_C=(double*)malloc(size);
 if(h_A==NULL||h_B==NULL||h_C==NULL) 	{
	 printf("malloc failed!");
	 exit(1);
 }
 for(i=0;i<N;i++)	{
 	h_A[i]=i*2;
 	h_B[i]=i*3;
 }
 vectorAdd(h_A,h_B,h_C,N);
 for(i=0;i<10;i++) {
 	printf("h_C[%d] is %f,should be %f\n",i,h_C[i],h_A[i]+h_B[i]);
 }
 return 0;
}

void vectorAdd(double* A, double* B,double* C,int n)	{
	double *d_A=NULL, *d_B=NULL, *d_C=NULL;
	int size=sizeof(double)*n;
	hipMalloc((void**)&d_A,size);
	hipMemcpy(d_A,A,size,hipMemcpyHostToDevice);
	hipMalloc((void**)&d_B,size);
	hipMemcpy(d_B,B,size,hipMemcpyHostToDevice);
	hipMalloc((void**)&d_C,size);
	if(d_A==NULL||d_B==NULL||d_C==NULL){	
		printf("device allocate memory failed!\n");
	}
//	dim3 dimGrid(65537,65537,65537);//test grid,block size
//	dim3 dimBlock(1026,1024,64);
	vecAddKernel<<<ceil(n/1024.0),1024>>>(d_A,d_B,d_C,n);

	hipMemcpy(C,d_C,size,hipMemcpyDeviceToHost);
	hipFree(d_A);hipFree(d_B);hipFree(d_C);
}

__global__ void vecAddKernel(double* A, double* B, double* C, int n)	{
	int i=blockDim.x*blockIdx.x+threadIdx.x;
	if(i<n) {
		C[i]=A[i]+B[i];
	}
}

//add nvcc -arch compute_13 to enable double
/*
Device 0: "Quadro K600"
  CUDA Driver Version / Runtime Version          5.5 / 5.5
  CUDA Capability Major/Minor version number:    3.0
  Total amount of global memory:                 1024 MBytes (1073414144 bytes)
  ( 1) Multiprocessors, (192) CUDA Cores/MP:     192 CUDA Cores
  GPU Clock rate:                                876 MHz (0.88 GHz)
  Memory Clock rate:                             891 Mhz
  Memory Bus Width:                              128-bit
  L2 Cache Size:                                 262144 bytes
  Maximum Texture Dimension Size (x,y,z)         1D=(65536), 2D=(65536, 65536), 3D=(4096, 4096, 4096)
  Maximum Layered 1D Texture Size, (num) layers  1D=(16384), 2048 layers
  Maximum Layered 2D Texture Size, (num) layers  2D=(16384, 16384), 2048 layers
  Total amount of constant memory:               65536 bytes
  Total amount of shared memory per block:       49152 bytes
  Total number of registers available per block: 65536
  Warp size:                                     32
  Maximum number of threads per multiprocessor:  2048
  Maximum number of threads per block:           1024
  Max dimension size of a thread block (x,y,z): (1024, 1024, 64)
  Max dimension size of a grid size    (x,y,z): (2147483647, 65535, 65535)
  Maximum memory pitch:                          2147483647 bytes
  Texture alignment:                             512 bytes
  Concurrent copy and kernel execution:          Yes with 1 copy engine(s)
  Run time limit on kernels:                     Yes
  Integrated GPU sharing Host Memory:            No
  Support host page-locked memory mapping:       Yes
  Alignment requirement for Surfaces:            Yes
  Device has ECC support:                        Disabled
  Device supports Unified Addressing (UVA):      Yes
  Device PCI Bus ID / PCI location ID:           5 / 0
  Compute Mode:
     < Default (multiple host threads can use ::cudaSetDevice() with device simultaneously) >

deviceQuery, CUDA Driver = CUDART, CUDA Driver Version = 5.5, CUDA Runtime Version = 5.5, NumDevs = 1, Device0 = Quadro K600
Result = PASS
*/
