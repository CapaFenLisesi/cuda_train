#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void pictureKernel(float* d_pix,int X, int Y);
int main() {
	float *h_pixin, *h_pixout, *d_pix;
	int x=76,y=62,i;
	int size=x*y*sizeof(float);
	dim3 dim_block(16,16,1);
	dim3 dim_grid(ceil(x/16.),ceil(y/16.),1);

	h_pixin=(float*)malloc(size);
	h_pixout=(float*)malloc(size);
	hipMalloc((void**) &d_pix,size);
	if(h_pixin==NULL||h_pixout==NULL||d_pix==NULL)	{
		printf("malloc failed!\n");
	}
	for(i=0;i<x*y;i++)	{
		h_pixin[i]=i*1.5;
	}
	hipMemcpy(d_pix,h_pixin,size,hipMemcpyHostToDevice);
	pictureKernel<<<dim_grid,dim_block>>>(d_pix,x,y);//after this d_pix changed
	hipMemcpy(h_pixout,d_pix,size,hipMemcpyDeviceToHost);
//	printf("gridDim.x=%d,gridDim.y=%d,blockDim.x=%d,blockDim.y=%d\n",gridDim.x,gridDim.y,blockDim.x,blockDim.y);
//these varibles need to access in kernel function
	for(i=0;i<x*y;i++)	{
		printf("h_pixin[i]=%f,h_pixout=%f\n",h_pixin[i],h_pixout[i]);
	}
	return 0;
}

__global__ void pictureKernel(float* d_pix,int X, int Y) {
	int thread_x=blockDim.x*blockIdx.x+threadIdx.x;
	int thread_y=blockDim.y*blockIdx.y+threadIdx.y;
	if(thread_x<X&&thread_y<Y)	{
		d_pix[thread_y*X+thread_x]*=2;
	}
}
